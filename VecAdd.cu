
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdlib.h>

enum {
    grid_count=16
};
__global__ void vectorAdditionKernel(float * A , float * B , float * C ,int dataCount){
    int index = blockIdx.x *blockDim.x + threadIdx.x;
    if(index < dataCount) 
        C[index] = A[index] + B[index];
}

int main(){
    int dataCount = 2100;
    float h_A[dataCount];
    float h_B[dataCount];
    float h_C[dataCount];

// initialize the values 

    for(int i = 0 ; i < dataCount ; i++){

        h_A[i] = (float) i ;
        h_B[i] = (float) i ;
    }

    float * d_A;
    hipMalloc(&d_A , dataCount * sizeof(float));
    hipMemcpy(d_A ,h_A ,sizeof(float) * dataCount ,  hipMemcpyHostToDevice );

    float * d_B;
    hipMalloc(&d_B , dataCount * sizeof(float));
    hipMemcpy(d_B , h_B , sizeof(float) * dataCount , hipMemcpyHostToDevice);

    float * d_C;
    hipMalloc(&d_C , dataCount * sizeof(float));
    hipMemcpy(d_C , h_C , dataCount * sizeof(float) , hipMemcpyHostToDevice);

    // call the kernel
    int threadPerBlock = dataCount/grid_count;
    vectorAdditionKernel<<<grid_count,threadPerBlock>>> (d_A, d_B , d_C, dataCount);

    // get the data
    hipMemcpy(h_C , d_C , dataCount * sizeof(float) , hipMemcpyDeviceToHost);

    for(int i = 0 ; i < dataCount ;i++){
        printf("%f \n" , h_C[i]);

    }
    return 0 ;
    
}